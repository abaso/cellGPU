#include "hip/hip_runtime.h"
#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "vertexModelBase.cuh"

/** \file simpleVertexModelBase.cu
    * Defines kernel callers and kernels for GPU calculations of vertex models
*/

/*!
    \addtogroup vmKernels
    @{
*/

/*!
  This function is being deprecated, but is still useful for calculating, e.g. the mean-squared
displacement of the cells without transferring data back to the host
*/
__global__ void vm_get_cell_positions_kernel(Dscalar2* d_cellPositions,
                                              Dscalar2* d_vertexPositions,
                                              int    * d_nn,
                                              int    * d_n,
                                              int N,
                                              Index2D n_idx,
                                              gpubox Box)
    {
    // read in the cell index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;

    Dscalar2 vertex, pos, baseVertex;
    pos.x=0.0;pos.y=0.0;
    baseVertex = d_vertexPositions[ d_n[n_idx(0,idx)] ];
    int neighs = d_nn[idx];
    for (int n = 1; n < neighs; ++n)
        {
        Box.minDist(d_vertexPositions[ d_n[n_idx(n,idx)] ],baseVertex,vertex);
        pos.x += vertex.x;
        pos.y += vertex.y;
        };
    pos.x /= neighs;
    pos.y /= neighs;
    pos.x += baseVertex.x;
    pos.y += baseVertex.y;
    Box.putInBoxReal(pos);
    d_cellPositions[idx] = pos;
    };


//!Call the kernel to calculate the position of each cell from the position of its vertices
bool gpu_vm_get_cell_positions(
                    Dscalar2 *d_cellPositions,
                    Dscalar2 *d_vertexPositions,
                    int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    int      N,
                    Index2D  &n_idx,
                    gpubox   &Box)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    vm_get_cell_positions_kernel<<<nblocks,block_size>>>(d_cellPositions,d_vertexPositions,
                                                          d_cellVertexNum,d_cellVertices,
                                                          N, n_idx, Box);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
