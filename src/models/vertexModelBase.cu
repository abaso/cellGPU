#include "hip/hip_runtime.h"
#define NVCC
#define ENABLE_CUDA

#include <hip/hip_runtime.h>
#include "vertexModelBase.cuh"

/** \file vertexModelBase.cu
    * Defines kernel callers and kernels for GPU calculations of vertex models
*/

/*!
    \addtogroup vmKernels
    @{
*/

/*!
  Since the cells are NOT guaranteed to be convex, the area of the cell must take into account any
  self-intersections. The strategy is the same as in the CPU branch.
  */
__global__ void vm_geometry_kernel(
                                   const Dscalar2* __restrict__ d_vertexPositions,
                                   const int*  __restrict__ d_cellVertexNum,
                                   const int*  __restrict__ d_cellVertices,
                                   const int*  __restrict__ d_vertexCellNeighbors,
                                   Dscalar2*  __restrict__ d_voroCur,
                                   Dscalar4*  __restrict__ d_voroLastNext,
                                   Dscalar2*  __restrict__ d_AreaPerimeter,
                                   int N,
                                   Index2D cellNeighborIndexer,
                                   gpubox Box
                                    )
    {
    // read in the cell index that belongs to this thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= N)
        return;
    int neighs = d_cellVertexNum[idx];
    //Define the vertices of a cell relative to some (any one ) of its vertices to take care of periodic BCs
    Dscalar2 cellPos = d_vertexPositions[ d_cellVertices[cellNeighborIndexer(neighs-2,idx)]];
    Dscalar2 vlast, vcur,vnext;
    Dscalar Varea = 0.0;
    Dscalar Vperi = 0.0;

    vlast.x = 0.0; vlast.y=0.0;
    int vidx = d_cellVertices[cellNeighborIndexer(neighs-1,idx)];
    Box.minDist(d_vertexPositions[vidx],cellPos,vcur);
    for (int nn = 0; nn < neighs; ++nn)
        {
        //for easy force calculation, save the current, last, and next voronoi vertex position
        //in the approprate spot.
        int forceSetIdx = -1;
        for (int ff = 0; ff < 3; ++ff)
            {
            if(forceSetIdx != -1) continue;
            if(d_vertexCellNeighbors[3*vidx+ff]==idx)
                forceSetIdx = 3*vidx+ff;
            };

        vidx = d_cellVertices[cellNeighborIndexer(nn,idx)];
        Box.minDist(d_vertexPositions[vidx],cellPos,vnext);

        //compute area contribution. It is
        // 0.5 * (vcur.x+vnext.x)*(vnext.y-vcur.y)
        Varea += SignedPolygonAreaPart(vcur,vnext);
        Dscalar dx = vcur.x-vnext.x;
        Dscalar dy = vcur.y-vnext.y;
        Vperi += sqrt(dx*dx+dy*dy);
        //save voronoi positions in a convenient form
        d_voroCur[forceSetIdx] = vcur;
        d_voroLastNext[forceSetIdx] = make_Dscalar4(vlast.x,vlast.y,vnext.x,vnext.y);
        //advance the loop
        vlast = vcur;
        vcur = vnext;
        };
    d_AreaPerimeter[idx].x=Varea;
    d_AreaPerimeter[idx].y=Vperi;
    };

/*!
  Run through every pair of vertices (once), see if any T1 transitions should be done,
  and see if the cell-vertex list needs to grow
  */
__global__ void vm_simple_T1_test_kernel(Dscalar2* d_vertexPositions,
                                        int      *d_vertexNeighbors,
                                        int      *d_vertexEdgeFlips,
                                        int      *d_vertexCellNeighbors,
                                        int      *d_cellVertexNum,
                                        gpubox   Box,
                                        Dscalar  T1THRESHOLD,
                                        int      NvTimes3,
                                        int      vertexMax,
                                        int      *d_grow)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= NvTimes3)
        return;
    int vertex1 = idx/3;
    int vertex2 = d_vertexNeighbors[idx];
    Dscalar2 edge;
    if(vertex1 < vertex2)
        {
        Box.minDist(d_vertexPositions[vertex1],d_vertexPositions[vertex2],edge);
        if(norm(edge) < T1THRESHOLD)
            {
            d_vertexEdgeFlips[idx]=1;


            //test the number of neighbors of the cells connected to v1 and v2 to see if the
            //cell list should grow. This is kind of slow, and I wish I could optimize it away,
            //or at least not test for it during every time step. The latter seems pretty doable.
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex1+1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex1+2]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex2]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex2+1]] == vertexMax)
                d_grow[0] = 1;
            if(d_cellVertexNum[d_vertexCellNeighbors[3*vertex2+2]] == vertexMax)
                d_grow[0] = 1;
            }
        else
            d_vertexEdgeFlips[idx]=0;
        }
    else
        d_vertexEdgeFlips[idx] = 0;
    };

/*!
  There will be severe topology mismatches if a cell is involved in more than one T1 transition
  simultaneously (due to incoherent updates of the cellVertices structure). So, go through the
  current list of edges that are marked to take part in a T1 transition and select one edge per
  cell to be flipped on this trip through the functions.
  */
__global__ void vm_one_T1_per_cell_per_vertex_kernel(
                                        int* __restrict__ d_vertexEdgeFlips,
                                        int* __restrict__ d_vertexEdgeFlipsCurrent,
                                        const int* __restrict__ d_vertexNeighbors,
                                        const int* __restrict__ d_vertexCellNeighbors,
                                        const int* __restrict__ d_cellVertexNum,
                                        const int * __restrict__ d_cellVertices,
                                        int *d_finishedFlippingEdges,
                                        Index2D cellNeighborIndexer,
                                        int Ncells)
    {
    unsigned int cell = blockDim.x * blockIdx.x + threadIdx.x;
    if (cell >= Ncells)
        return;

    //look through every vertex of the cell
    int cneigh = d_cellVertexNum[cell];
    int vertex;
    bool flipFound = false;
    bool moreFlipsFound = false;
    for (int cc = 0; cc < cneigh; ++cc)
        {
        vertex = d_cellVertices[cellNeighborIndexer(cc,cell)];
        //what are the other cells attached to this vertex? For correctness, only one cell should
        //own each vertex here. For simplicity, only the lowest-indexed cell gets to do any work.
        if(d_vertexCellNeighbors[3*vertex] < cell ||
               d_vertexCellNeighbors[3*vertex+1] < cell ||
               d_vertexCellNeighbors[3*vertex+2] < cell)
            continue;

        if(d_vertexEdgeFlips[3*vertex] == 1)
            {
            d_vertexEdgeFlipsCurrent[3*vertex] = 1;
            d_vertexEdgeFlips[3*vertex] = 0;
            if (flipFound)
                {
                moreFlipsFound = true;
                break;
                }
            flipFound = true;
            };
        if(d_vertexEdgeFlips[3*vertex+1] == 1)
            {
            d_vertexEdgeFlipsCurrent[3*vertex+1] = 1;
            d_vertexEdgeFlips[3*vertex+1] = 0;
            if (flipFound)
                {
                moreFlipsFound = true;
                break;
                }
            flipFound = true;
            };
        if(d_vertexEdgeFlips[3*vertex+2] == 1)
            {
            d_vertexEdgeFlipsCurrent[3*vertex+2] = 1;
            d_vertexEdgeFlips[3*vertex+2] = 0;
            if (flipFound)
                {
                moreFlipsFound = true;
                break;
                }
            flipFound = true;
            };
        };
    if (flipFound)
        {
        d_finishedFlippingEdges[0] = 1;
        if(moreFlipsFound)
            d_finishedFlippingEdges[1] = 1;
        };
    };

/*!
  Flip any edge labeled for re-wiring in the vertexEdgeFlipsCurrent list
  */
__global__ void vm_flip_edges_kernel(int* d_vertexEdgeFlipsCurrent,
                                      Dscalar2 *d_vertexPositions,
                                      int      *d_vertexNeighbors,
                                      int      *d_vertexCellNeighbors,
                                      int      *d_cellVertexNum,
                                      int      *d_cellVertices,
                                      Dscalar  T1Threshold,
                                      gpubox   Box,
                                      Index2D  cellNeighborIndexer,
                                      int      NvTimes3)
    {
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //return if the index is out of bounds or if the edge isn't marked for flipping
    if (idx >= NvTimes3 || d_vertexEdgeFlipsCurrent[idx] == 0)
        return;
    //identify the vertices and reset the flag
    int vertex1 = idx/3;
    int vertex2 = d_vertexNeighbors[idx];
    d_vertexEdgeFlipsCurrent[idx] = 0;

    //Rotate the vertices in the edge and set them at twice their original distance
    Dscalar2 edge;
    Dscalar2 v1 = d_vertexPositions[vertex1];
    Dscalar2 v2 = d_vertexPositions[vertex2];
    Box.minDist(v1,v2,edge);
    if(norm(edge) < T1Threshold) return;

    //Dscalar2 midpoint;
    //midpoint.x = v2.x + 0.5*edge.x;
    //midpoint.y = v2.y + 0.5*edge.y;

    //v1.x = midpoint.x-edge.y;v1.y = midpoint.y+edge.x;
    //v2.x = midpoint.x+edge.y;v2.y = midpoint.y-edge.x;
    v1.x = v2.x + 0.5*edge.x-edge.y;
    v1.y = v2.y + 0.5*edge.y+edge.x;
    v2.x = v2.x + 0.5*edge.x+edge.y;
    v2.y = v2.y + 0.5*edge.y-edge.x;
    Box.putInBoxReal(v1);
    Box.putInBoxReal(v2);
    d_vertexPositions[vertex1] = v1;
    d_vertexPositions[vertex2] = v2;

    //now, do the gross work of cell and vertex rewiring
    int4 cellSet;cellSet.x=-1;cellSet.y=-1;cellSet.z=-1;cellSet.w=-1;
    //int4 vertexSet;
    int2 vertexSet;
    /*
    The following is fairly terrible GPU code, and should be considered for refactoring
    On the other hand, revising or improving the multiple-call structure of the edge-flipping
    routine would be a much large optimization
    */
    int cell1,cell2,cell3,ctest;
    int vlast, vcur, vnext, cneigh;
    cell1 = d_vertexCellNeighbors[3*vertex1];
    cell2 = d_vertexCellNeighbors[3*vertex1+1];
    cell3 = d_vertexCellNeighbors[3*vertex1+2];
    //cell_l doesn't contain vertex 1, so it is the cell neighbor of vertex 2 we haven't found yet
    for (int ff = 0; ff < 3; ++ff)
        {
        ctest = d_vertexCellNeighbors[3*vertex2+ff];
        if(ctest != cell1 && ctest != cell2 && ctest != cell3)
            cellSet.w=ctest;
        };
    //find vertices "c" and "d"
    cneigh = d_cellVertexNum[cellSet.w];
    vlast = d_cellVertices[ cellNeighborIndexer(cneigh-2,cellSet.w) ];
    vcur = d_cellVertices[ cellNeighborIndexer(cneigh-1,cellSet.w) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[cellNeighborIndexer(cn,cell1)];
        if(vcur == vertex2) break;
        vlast = vcur;
        vcur = vnext;
        };

    //classify cell1
    cneigh = d_cellVertexNum[cell1];
    vlast = d_cellVertices[ cellNeighborIndexer(cneigh-2,cell1) ];
    vcur = d_cellVertices[ cellNeighborIndexer(cneigh-1,cell1) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[cellNeighborIndexer(cn,cell1)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    if(vlast == vertex2)
        cellSet.x = cell1;
    else if(vnext == vertex2)
        cellSet.z = cell1;
    else
        {
        cellSet.y = cell1;
        };

    //classify cell2
    cneigh = d_cellVertexNum[cell2];
    vlast = d_cellVertices[ cellNeighborIndexer(cneigh-2,cell2) ];
    vcur = d_cellVertices[ cellNeighborIndexer(cneigh-1,cell2) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[cellNeighborIndexer(cn,cell2)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    if(vlast == vertex2)
        cellSet.x = cell2;
    else if(vnext == vertex2)
        cellSet.z = cell2;
    else
        {
        cellSet.y = cell2;
        };

    //classify cell3
    cneigh = d_cellVertexNum[cell3];
    vlast = d_cellVertices[ cellNeighborIndexer(cneigh-2,cell3) ];
    vcur = d_cellVertices[ cellNeighborIndexer(cneigh-1,cell3) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[cellNeighborIndexer(cn,cell3)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    if(vlast == vertex2)
        cellSet.x = cell3;
    else if(vnext == vertex2)
        cellSet.z = cell3;
    else
        {
        cellSet.y = cell3;
        };

    //get the vertexSet by examining cells j and l
    cneigh = d_cellVertexNum[cellSet.y];
    vlast = d_cellVertices[ cellNeighborIndexer(cneigh-2,cellSet.y) ];
    vcur = d_cellVertices[ cellNeighborIndexer(cneigh-1,cellSet.y) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[cellNeighborIndexer(cn,cellSet.y)];
        if(vcur == vertex1) break;
        vlast = vcur;
        vcur = vnext;
        };
    //vertexSet.x=vlast;
    //vertexSet.y=vnext;
    vertexSet.x=vnext;
    cneigh = d_cellVertexNum[cellSet.w];
    vlast = d_cellVertices[ cellNeighborIndexer(cneigh-2,cellSet.w) ];
    vcur = d_cellVertices[ cellNeighborIndexer(cneigh-1,cellSet.w) ];
    for (int cn = 0; cn < cneigh; ++cn)
        {
        vnext = d_cellVertices[cellNeighborIndexer(cn,cellSet.w)];
        if(vcur == vertex2) break;
        vlast = vcur;
        vcur = vnext;
        };
    //vertexSet.w=vlast;
    //vertexSet.z=vnext;
    vertexSet.y=vnext;

    /*
    Great, that was the first chunk of terrible code... but the nightmare isn't over
    */

    //re-wire the cells and vertices
    //start with the vertex-vertex and vertex-cell  neighbors
    for (int vert = 0; vert < 3; ++vert)
        {
        //vertex-cell neighbors
        if(d_vertexCellNeighbors[3*vertex1+vert] == cellSet.z)
            d_vertexCellNeighbors[3*vertex1+vert] = cellSet.w;
        if(d_vertexCellNeighbors[3*vertex2+vert] == cellSet.x)
            d_vertexCellNeighbors[3*vertex2+vert] = cellSet.y;
        //vertex-vertex neighbors
        if(d_vertexNeighbors[3*vertexSet.x+vert] == vertex1)
            d_vertexNeighbors[3*vertexSet.x+vert] = vertex2;
        if(d_vertexNeighbors[3*vertexSet.y+vert] == vertex2)
            d_vertexNeighbors[3*vertexSet.y+vert] = vertex1;
        if(d_vertexNeighbors[3*vertex1+vert] == vertexSet.x)
            d_vertexNeighbors[3*vertex1+vert] = vertexSet.y;
        if(d_vertexNeighbors[3*vertex2+vert] == vertexSet.y)
            d_vertexNeighbors[3*vertex2+vert] = vertexSet.x;
        };

    //now rewire the cells...
    //cell i loses v2 as a neighbor
    cneigh = d_cellVertexNum[cellSet.x];
    int cidx = 0;
    for (int cc = 0; cc < cneigh-1; ++cc)
        {
        if(d_cellVertices[cellNeighborIndexer(cc,cellSet.x)] == vertex2)
            cidx +=1;
        d_cellVertices[cellNeighborIndexer(cc,cellSet.x)] = d_cellVertices[cellNeighborIndexer(cidx,cellSet.x)];
        cidx +=1;
        };
    d_cellVertexNum[cellSet.x] -= 1;

    //cell j gains v2 in between v1 and b, so step through list backwards and insert
    cneigh = d_cellVertexNum[cellSet.y];
    cidx = cneigh;
    int vLocation = cneigh;
    for (int cc = cneigh-1;cc >=0; --cc)
        {
        int cellIndex = d_cellVertices[cellNeighborIndexer(cc,cellSet.y)];
        if(cellIndex == vertex1)
            {
            vLocation = cidx;
            cidx -= 1;
            };
        d_cellVertices[cellNeighborIndexer(cidx,cellSet.y)] = cellIndex;
        cidx -= 1;
        };
    if(cidx ==0)
        d_cellVertices[cellNeighborIndexer(0,cellSet.y)] = vertex2;
    else
        d_cellVertices[cellNeighborIndexer(vLocation,cellSet.y)] = vertex2;
    d_cellVertexNum[cellSet.y] += 1;

    //cell k loses v1 as a neighbor
    cneigh = d_cellVertexNum[cellSet.z];
    cidx = 0;
    for (int cc = 0; cc < cneigh-1; ++cc)
        {
        if(d_cellVertices[cellNeighborIndexer(cc,cellSet.z)] == vertex1)
            cidx +=1;
        d_cellVertices[cellNeighborIndexer(cc,cellSet.z)] = d_cellVertices[cellNeighborIndexer(cidx,cellSet.z)];
        cidx +=1;
        };
    d_cellVertexNum[cellSet.z] -= 1;

    //cell l gains v1 in between v2 and c...copy the logic of cell j
    cneigh = d_cellVertexNum[cellSet.w];
    cidx = cneigh;
    vLocation = cneigh;
    for (int cc = cneigh-1;cc >=0; --cc)
        {
        int cellIndex = d_cellVertices[cellNeighborIndexer(cc,cellSet.w)];
        if(cellIndex == vertex2)
            {
            vLocation = cidx;
            cidx -= 1;
            };
        d_cellVertices[cellNeighborIndexer(cidx,cellSet.w)] = cellIndex;
        cidx -= 1;
        };
    if(cidx ==0)
        d_cellVertices[cellNeighborIndexer(0,cellSet.w)] = vertex1;
    else
        d_cellVertices[cellNeighborIndexer(vLocation,cellSet.w)] = vertex1;
    d_cellVertexNum[cellSet.w] += 1;
    };

//!Call the kernel to calculate the area and perimeter of each cell
bool gpu_vm_geometry(
                    Dscalar2 *d_vertexPositions,
                    int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    int      *d_vertexCellNeighbors,
                    Dscalar2 *d_voroCur,
                    Dscalar4 *d_voroLastNext,
                    Dscalar2 *d_AreaPerimeter,
                    int      N,
                    Index2D  &cellNeighborIndexer,
                    gpubox   &Box)
    {
    unsigned int block_size = 128;
    if (N < 128) block_size = 32;
    unsigned int nblocks  = N/block_size + 1;


    vm_geometry_kernel<<<nblocks,block_size>>>(d_vertexPositions,
                                               d_cellVertexNum,d_cellVertices,
                                               d_vertexCellNeighbors,d_voroCur,
                                               d_voroLastNext,d_AreaPerimeter,
                                               N, cellNeighborIndexer, Box);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

//!Call the kernel to test every edge for a T1 event, see if vertexMax needs to increase
bool gpu_vm_test_edges_for_T1(
                    Dscalar2 *d_vertexPositions,
                    int      *d_vertexNeighbors,
                    int      *d_vertexEdgeFlips,
                    int      *d_vertexCellNeighbors,
                    int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    gpubox   &Box,
                    Dscalar  T1THRESHOLD,
                    int      Nvertices,
                    int      vertexMax,
                    int      *d_grow,
                    Index2D  &cellNeighborIndexer)
    {
    unsigned int block_size = 128;
    int NvTimes3 = Nvertices*3;
    if (NvTimes3 < 128) block_size = 32;
    unsigned int nblocks  = NvTimes3/block_size + 1;

    //test edges
    vm_simple_T1_test_kernel<<<nblocks,block_size>>>(
                                                      d_vertexPositions,d_vertexNeighbors,
                                                      d_vertexEdgeFlips,d_vertexCellNeighbors,
                                                      d_cellVertexNum,
                                                      Box,T1THRESHOLD,
                                                      NvTimes3,vertexMax,d_grow);

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };


//!determine whether any edges need to be flipped, and if we need to loop through the flipping routine, writing to d_finishedFlippingEdges the current state
bool gpu_vm_parse_multiple_flips(
                    int      *d_vertexEdgeFlips,
                    int      *d_vertexEdgeFlipsCurrent,
                    int      *d_vertexNeighbors,
                    int      *d_vertexCellNeighbors,
                    int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    int      *d_finishedFlippingEdges,
                    Index2D  &cellNeighborIndexer,
                    int      Ncells)
    {
    unsigned int block_size = 128;

    /*The issue is that if a cell is involved in two edge flips done by different threads, the resulting
    data structure for what vertices belong to cells and what cells border which vertex will be
    inconsistently updated.

    The strategy will be to take the d_vertexEdgeFlips list, put at most one T1 per cell per vertex into the
    d_vertexEdgeFlipsCurrent list (erasing it from the d_vertexEdgeFlips list), and swap the edges specified
    by the "current" list. If d_vertexEdgeFlips is empty, we will set d_finishedFlippingEdges[0] to 1,
     and if any cell has multiple edges to flip, we set d_finishedFlippingEdges[1] to 1. As long
    as the zeroth entry is 1, the flip edges kernel is called; as long as the first entry is 1 the cpp code will continue calling this gpu_avm_flip_edges function.
    */

    //first select a few edges to flip...
    if(Ncells <128) block_size = 32;
    unsigned int nblocks = Ncells/block_size + 1;
    vm_one_T1_per_cell_per_vertex_kernel<<<nblocks,block_size>>>(
                                                                d_vertexEdgeFlips,
                                                                d_vertexEdgeFlipsCurrent,
                                                                d_vertexNeighbors,
                                                                d_vertexCellNeighbors,
                                                                d_cellVertexNum,
                                                                d_cellVertices,
                                                                d_finishedFlippingEdges,
                                                                cellNeighborIndexer,
                                                                Ncells);
    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };


//!Call the kernel to flip at most one edge per cell, write to d_finishedFlippingEdges the current state
bool gpu_vm_flip_edges(
                    int      *d_vertexEdgeFlipsCurrent,
                    Dscalar2 *d_vertexPositions,
                    int      *d_vertexNeighbors,
                    int      *d_vertexCellNeighbors,
                    int      *d_cellVertexNum,
                    int      *d_cellVertices,
                    Dscalar  T1Threshold,
                    gpubox   &Box,
                    Index2D  &cellNeighborIndexer,
                    int      Nvertices,
                    int      Ncells)
    {
    unsigned int block_size = 128;

    if(Ncells <128) block_size = 32;

    int NvTimes3 = Nvertices*3;
    if (NvTimes3 < 128) block_size = 32;
    unsigned int nblocks  = NvTimes3/block_size + 1;

    vm_flip_edges_kernel<<<nblocks,block_size>>>(
                                                  d_vertexEdgeFlipsCurrent,d_vertexPositions,d_vertexNeighbors,
                                                  d_vertexCellNeighbors,d_cellVertexNum,d_cellVertices,
                                                  T1Threshold,Box,
                                                  cellNeighborIndexer,NvTimes3);

    HANDLE_ERROR(hipGetLastError());
    return hipSuccess;
    };

/** @} */ //end of group declaration
